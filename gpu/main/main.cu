#include "hip/hip_runtime.h"
//
// Created by will on 2020-03-23.
//

#include <cxxopts.hpp>
#include <fstream>

#include "gdbscan.cuh"

int main(int argc, char *argv[]) {
  cxxopts::Options options("GDBSCAN", "ma, look, it's GDBSCAN");
  // clang-format off
    options.add_options()
            ("p,print", "Print clustering IDs") // boolean
            ("r,eps", "Clustering radius", cxxopts::value<float>())
            ("n,min-samples", "Number of points within radius", cxxopts::value<size_t>())
            ("i,input", "Input filename", cxxopts::value<std::string>());
  // clang-format on
  auto args = options.parse(argc, argv);

  bool output_labels = args["print"].as<bool>();
  float radius = args["eps"].as<float>();
  uint min_pts = args["min-samples"].as<size_t>();
  std::string input = args["input"].as<std::string>();

  std::cout << "minPts=" << min_pts << "; eps=" << radius << std::endl;

  GDBSCAN::Solver solver(input, min_pts, radius);

  solver.calc_num_neighbours();
  solver.calc_start_pos();
  solver.append_neighbours();
  solver.identify_cores();
  solver.identify_clusters();

  if (output_labels) {
    std::cout << "cluster ids:" << std::endl;
    for (const auto &id : solver.cluster_ids) std::cout << id << std::endl;
  }

  return 0;
}